#include "hip/hip_runtime.h"
/*
 * BaselineRemover_GPU.cpp
 *
 *  Created on: Aug 17, 2016
 *      Author: jlippuner
 */

#include "BaselineRemover.hpp"

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-parameter"
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/system/cuda/execution_policy.h>
#pragma GCC diagnostic pop

#include "CUDA.hpp"
#include "utils.hpp"

struct high_pass_functor {
  high_pass_functor(const float df, const float f_cutoff, const float div) :
      _df(df),
      _f_cutoff(f_cutoff),
      _div(div) {
  }

  __device__ hipfftComplex operator()(const hipfftComplex val, const size_t idx) {
    float f = (float)idx * _df;
    float mult = _div * 0.5 * (tanhf(2.0 * (f - _f_cutoff)) + 1.0);

    hipfftComplex res = val;
    res.x = val.x * mult;
    res.y = val.y * mult;
    return res;
  }

  const float _df;
  const float _f_cutoff;
  const float _div;
};

struct GPU_Impl : public Impl {
  GPU_Impl(const size_t stream_batch_size, const size_t N_pad, const size_t num_out,
      const size_t out_size, const int real_stride, const int complex_stride) :
      Stream_batch_size(stream_batch_size),
      d_data(nullptr),
      d_real(new hipfftReal*[Stream_batch_size]),
      d_complex(new hipfftComplex*[Stream_batch_size]),
      stream(new hipStream_t[Stream_batch_size]),
      plan_r2c(new hipfftHandle[Stream_batch_size]),
      plan_c2r(new hipfftHandle[Stream_batch_size]) {
    CUCHK(hipMalloc(&d_data, Stream_batch_size * out_size));

    int N_pad_int = (int)N_pad;
    int num_out_int = (int)num_out;

    for (size_t s = 0; s < Stream_batch_size; ++s) {
      d_real[s] = d_data + s * real_stride;
      d_complex[s] = (hipfftComplex*)d_real[s];

      CUCHK(hipStreamCreate(stream + s));
      CUFFTCHK(hipfftPlanMany(plan_r2c + s, 1, &N_pad_int, &N_pad_int, 1,
          real_stride, &num_out_int, 1, complex_stride, HIPFFT_R2C, 1));
      CUFFTCHK(hipfftPlanMany(plan_c2r + s, 1, &N_pad_int, &num_out_int, 1,
          complex_stride, &N_pad_int, 1, real_stride, HIPFFT_C2R, 1));

      CUFFTCHK(hipfftSetStream(plan_r2c[s], stream[s]));
      CUFFTCHK(hipfftSetStream(plan_c2r[s], stream[s]));
    }
  }

  ~GPU_Impl() {
    CUCHK(hipFree(d_data));

    for (size_t s = 0; s < Stream_batch_size; ++s) {
      CUFFTCHK(hipfftDestroy(plan_r2c[s]));
      CUFFTCHK(hipfftDestroy(plan_c2r[s]));
      CUCHK(hipStreamDestroy(stream[s]));
    }

    delete [] d_real;
    delete [] d_complex;
    delete [] stream;
    delete [] plan_r2c;
    delete [] plan_c2r;
  }

  size_t Stream_batch_size;

  hipfftReal * d_data;
  hipfftReal ** d_real;
  hipfftComplex ** d_complex;

  hipStream_t * stream;

  hipfftHandle * plan_r2c;
  hipfftHandle * plan_c2r;
};

bool BaselineRemover::GPU_Available() {
  return true;
}

void BaselineRemover::GPU_Init(const size_t total_num_channels) {
  // round up to multiple of 256 bytes
  mOut_size = 256 * ((mOut_size + 255) / 256);

  int real_stride = mOut_size / sizeof(hipfftReal);
  int complex_stride = mOut_size / sizeof(hipfftComplex);

  // figure out who many channels we can do in parallel
  size_t free_mem = 0;
  size_t total_mem = 0;
  CUCHK(hipMemGetInfo(&free_mem, &total_mem));

  size_t work_size_r2c, work_size_c2r;
  int N_pad_int = (int)mN_pad;
  int num_out_int = (int)mNum_out;

  CUFFTCHK(hipfftEstimateMany(1, &N_pad_int, &N_pad_int, 1, real_stride,
      &num_out_int, 1, complex_stride, HIPFFT_R2C, 1, &work_size_r2c));

  CUFFTCHK(hipfftEstimateMany(1, &N_pad_int, &num_out_int, 1, complex_stride,
      &N_pad_int, 1, real_stride, HIPFFT_C2R, 1, &work_size_c2r));

  size_t total_size = mOut_size + work_size_r2c + work_size_c2r;
  size_t stream_batch_size = 0.95 * (double)free_mem / (double)total_size;

  stream_batch_size = std::min(stream_batch_size, total_num_channels);
  stream_batch_size = std::min(stream_batch_size, (size_t)32);

  if (stream_batch_size <= 0)
    throw std::runtime_error("Not enough memory for SigProc::RemoveBaseline");

  mpImpl = new GPU_Impl(stream_batch_size, mN_pad, mNum_out, mOut_size,
      real_stride, complex_stride);
}

void BaselineRemover::GPU_Process_batch(float * const data,
    const size_t num_channels) {
  auto impl = dynamic_cast<GPU_Impl*>(mpImpl);
  if (impl == nullptr)
    throw std::runtime_error("Could not cast mpImpl to GPU_Impl");

  size_t num_stream_batches = (num_channels + impl->Stream_batch_size - 1)
      / impl->Stream_batch_size;

  for (size_t i = 0; i < num_stream_batches; ++i) {
    size_t num_streams = std::min(impl->Stream_batch_size,
        num_channels - i * impl->Stream_batch_size);

    for (size_t s = 0; s < num_streams; ++s) {
      size_t channel_idx = i * impl->Stream_batch_size + s;

      // copy data to GPU
      CUCHK(hipMemsetAsync(impl->d_real[s], 0, mOut_size,
          impl->stream[s]));
      CUCHK(hipMemcpyAsync(impl->d_real[s], data + channel_idx * mN,
          mN * sizeof(float), hipMemcpyHostToDevice, impl->stream[s]));

      // subtract mean of signal
      thrust::device_ptr<float> signal(impl->d_real[s]);
      float sum = thrust::reduce(thrust::cuda::par.on(impl->stream[s]),
          signal, signal + mN);
      float mean = sum / (float)mN;
      thrust::for_each(thrust::cuda::par.on(impl->stream[s]), signal,
          signal + mN, thrust::placeholders::_1 -= mean);

      // do FFT
      CUFFTCHK(hipfftExecR2C(impl->plan_r2c[s], impl->d_real[s],
          impl->d_complex[s]));

      // apply high pass filter
      thrust::device_ptr<hipfftComplex> spectrum(impl->d_complex[s]);
      thrust::counting_iterator<size_t> cnt(0);
      thrust::transform(thrust::cuda::par.on(impl->stream[s]), spectrum,
          spectrum + mNum_out, cnt, spectrum,
          high_pass_functor(mDf, mF_cutoff, mDiv));

      CUFFTCHK(hipfftExecC2R(impl->plan_c2r[s], impl->d_complex[s],
          impl->d_real[s]));

      // copy data back
      CUCHK(hipMemcpyAsync(data + channel_idx * mN, impl->d_real[s],
          mN * sizeof(float), hipMemcpyDeviceToHost, impl->stream[s]));
    }
  }

  for (size_t s = 0; s < impl->Stream_batch_size; ++s) {
    CUCHK(hipStreamSynchronize(impl->stream[s]));
  }
}

size_t BaselineRemover::GPU_Ram_per_channel() const {
  return 0;
}
